#include "hip/hip_runtime.h"
#include <cstdint>
#include <cassert>
#include <hip/hip_runtime.h>
#include <nba/engines/cuda/utils.hh>
#include <nba/core/errors.hh>
#include <nba/core/accumidx.hh>
#include <nba/framework/datablock_shared.hh>

#include "IPsecAuthHMACSHA1_kernel.hh"

/* The index is given by the order in get_used_datablocks(). */
#define dbid_enc_payloads_d (0)
#define dbid_flow_ids_d     (1)

#define SHA1_THREADS_PER_BLK 32

extern "C" {

//__global__ uint32_t d_pad_buffer[16 * 2 * MAX_CHUNK_SIZE * MAX_GROUP_SIZE];

__device__ static uint32_t swap(uint32_t v) {
    return ((v & 0x000000ffU) << 24) | ((v & 0x0000ff00U) << 8)
            | ((v & 0x00ff0000U) >> 8) | ((v & 0xff000000U) >> 24);
}

typedef struct hash_digest {
    uint32_t h1;
    uint32_t h2;
    uint32_t h3;
    uint32_t h4;
    uint32_t h5;
} hash_digest_t;

#define HMAC

__inline__ __device__ static void getBlock(char* buf, int offset, int len, uint32_t* dest)
{
    uint32_t *tmp;
    unsigned int tempbuf[16];

    tmp = (uint32_t*) (buf + offset);
    //printf("%d %d\n", offset, len);
    if (offset + 64 <= len) {
        //printf("--0--\n");
#pragma unroll 16
        for (int i = 0; i < 16; i++) {
            dest[i] = swap(tmp[i]);
        }
    } else if (len > offset && (len - offset) < 56) { //case 1 enough space in last block for padding
        //prtinf("--1--\n");
        int i;
        for (i = 0; i < (len - offset) / 4; i++) {

            //printf("%d %d\n",offset,i);
            //printf("%p %p\n", buf, dest);

            //tempbuf[i] = buf[i];
            tempbuf[i] = swap(tmp[i]);
        }
        //printf("len%%4 %d\n",len%4);
        switch (len % 4) {
        case 0:
            tempbuf[i] = swap(0x00000080);
            i++;
            break;
        case 1:
            tempbuf[i] = swap(0x00008000 | (tmp[i] & 0x000000FF));
            i++;
            break;
        case 2:
            tempbuf[i] = swap(0x00800000 | (tmp[i] & 0x0000FFFF));
            i++;
            break;
        case 3:
            tempbuf[i] = swap(0x80000000 | (tmp[i] & 0x00FFFFFF));
            i++;
            break;
        };
        for (; i < 14; i++) {
            tempbuf[i] = 0;
        }
#pragma unroll 14
        for (i = 0; i < 14; i++) {
            dest[i] = tempbuf[i];
        }
        dest[14] = 0x00000000;
#ifndef HMAC
        dest[15] = len * 8;
#else
        dest[15] = (len + 64) * 8;
#endif

    } else if (len > offset && (len - offset) >= 56) { //case 2 not enough space in last block (containing message) for padding
        //printf("--2--\n");
        int i;
        for (i = 0; i < (len - offset) / 4; i++) {
            tempbuf[i] = swap(tmp[i]);
        }
        switch (len % 4) {
        case 0:
            tempbuf[i] = swap(0x00000080);
            i++;
            break;
        case 1:
            tempbuf[i] = swap(0x00008000 | (tmp[i] & 0x000000FF));
            i++;
            break;
        case 2:
            tempbuf[i] = swap(0x00800000 | (tmp[i] & 0x0000FFFF));
            i++;
            break;
        case 3:
            tempbuf[i] = swap(0x80000000 | (tmp[i] & 0x00FFFFFF));
            i++;
            break;
        };

        for (; i < 16; i++) {
            tempbuf[i] = 0x00000000;
        }

#pragma unroll 16
        for (i = 0; i < 16; i++) {
            dest[i] = tempbuf[i];
        }

    } else if (offset == len) { //message end is aligned in 64 bytes
        //printf("--3--\n");
        dest[0] = swap(0x00000080);
#pragma unroll 13
        for (int i = 1; i < 14; i++)
            dest[i] = 0x00000000;
        dest[14] = 0x00000000;
#ifndef HMAC
        dest[15] = len * 8;
#else
        dest[15] = (len + 64) * 8;
#endif

    } else if (offset > len) { //the last block in case 2
        //printf("--4--\n");
#pragma unroll 14
        for (int i = 0; i < 14; i++)
            dest[i] = 0x00000000;
        dest[14] = 0x00000000;
#ifndef HMAC
        dest[15] = len * 8;
#else
        dest[15] = (len + 64) * 8;
#endif

    } else {
        printf("Not supposed to happen\n");
    }
}

__device__ static void computeSHA1Block(char* in, uint32_t* w, int offset, int len,
        hash_digest_t &h) {
    uint32_t a = h.h1;
    uint32_t b = h.h2;
    uint32_t c = h.h3;
    uint32_t d = h.h4;
    uint32_t e = h.h5;
    uint32_t f;
    uint32_t k;
    uint32_t temp;

    getBlock(in, offset, len, w);

    //for (int i = 0; i < 16 ; i++) {
    //  printf("%0X\n", w[i]);
    //}
    //printf("\n");

    k = 0x5A827999;
    //0 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
    w[0] = w[0] << 1 | w[0] >> 31;

    //1 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
    w[1] = w[1] << 1 | w[1] >> 31;

    //2 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
    w[2] = w[2] << 1 | w[2] >> 31;

    //3 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
    w[3] = w[3] << 1 | w[3] >> 31;

    //4 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
    w[4] = w[4] << 1 | w[4] >> 31;

    //5 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
    w[5] = w[5] << 1 | w[5] >> 31;

    //6 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
    w[6] = w[6] << 1 | w[6] >> 31;

    //7 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
    w[7] = w[7] << 1 | w[7] >> 31;

    //8 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
    w[8] = w[8] << 1 | w[8] >> 31;

    //9 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
    w[9] = w[9] << 1 | w[9] >> 31;

    //10 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
    w[10] = w[10] << 1 | w[10] >> 31;

    //11 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
    w[11] = w[11] << 1 | w[11] >> 31;

    //12 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
    w[12] = w[12] << 1 | w[12] >> 31;

    //13 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
    w[13] = w[13] << 1 | w[13] >> 31;

    //14 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
    w[14] = w[14] << 1 | w[14] >> 31;

    //15 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
    w[15] = w[15] << 1 | w[15] >> 31;

    //16 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
    w[0] = w[0] << 1 | w[0] >> 31;

    //17 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
    w[1] = w[1] << 1 | w[1] >> 31;

    //18 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
    w[2] = w[2] << 1 | w[2] >> 31;

    //19 of 0-20
    f = (b & c) | ((~b) & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
    w[3] = w[3] << 1 | w[3] >> 31;

    k = 0x6ED9EBA1;
    //20 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
    w[4] = w[4] << 1 | w[4] >> 31;

    //21 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
    w[5] = w[5] << 1 | w[5] >> 31;

    //22 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
    w[6] = w[6] << 1 | w[6] >> 31;

    //23 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
    w[7] = w[7] << 1 | w[7] >> 31;

    //24 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
    w[8] = w[8] << 1 | w[8] >> 31;

    //25 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
    w[9] = w[9] << 1 | w[9] >> 31;

    //26 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
    w[10] = w[10] << 1 | w[10] >> 31;

    //27 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
    w[11] = w[11] << 1 | w[11] >> 31;

    //28 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
    w[12] = w[12] << 1 | w[12] >> 31;

    //29 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
    w[13] = w[13] << 1 | w[13] >> 31;

    //30 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
    w[14] = w[14] << 1 | w[14] >> 31;

    //31 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
    w[15] = w[15] << 1 | w[15] >> 31;

    //32 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
    w[0] = w[0] << 1 | w[0] >> 31;

    //33 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
    w[1] = w[1] << 1 | w[1] >> 31;

    //34 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
    w[2] = w[2] << 1 | w[2] >> 31;

    //35 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
    w[3] = w[3] << 1 | w[3] >> 31;

    //36 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
    w[4] = w[4] << 1 | w[4] >> 31;

    //37 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
    w[5] = w[5] << 1 | w[5] >> 31;

    //38 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
    w[6] = w[6] << 1 | w[6] >> 31;

    //39 of 20-40
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
    w[7] = w[7] << 1 | w[7] >> 31;

    k = 0x8F1BBCDC;
    //40 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
    w[8] = w[8] << 1 | w[8] >> 31;

    //41 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
    w[9] = w[9] << 1 | w[9] >> 31;

    //42 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
    w[10] = w[10] << 1 | w[10] >> 31;

    //43 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
    w[11] = w[11] << 1 | w[11] >> 31;

    //44 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
    w[12] = w[12] << 1 | w[12] >> 31;

    //45 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
    w[13] = w[13] << 1 | w[13] >> 31;

    //46 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
    w[14] = w[14] << 1 | w[14] >> 31;

    //47 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
    w[15] = w[15] << 1 | w[15] >> 31;

    //48 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[0] = w[13] ^ w[8] ^ w[2] ^ w[0];
    w[0] = w[0] << 1 | w[0] >> 31;

    //49 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[1] = w[14] ^ w[9] ^ w[3] ^ w[1];
    w[1] = w[1] << 1 | w[1] >> 31;

    //50 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[2] = w[15] ^ w[10] ^ w[4] ^ w[2];
    w[2] = w[2] << 1 | w[2] >> 31;

    //51 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[3] = w[0] ^ w[11] ^ w[5] ^ w[3];
    w[3] = w[3] << 1 | w[3] >> 31;

    //52 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[4] = w[1] ^ w[12] ^ w[6] ^ w[4];
    w[4] = w[4] << 1 | w[4] >> 31;

    //53 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[5] = w[2] ^ w[13] ^ w[7] ^ w[5];
    w[5] = w[5] << 1 | w[5] >> 31;

    //54 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[6] = w[3] ^ w[14] ^ w[8] ^ w[6];
    w[6] = w[6] << 1 | w[6] >> 31;

    //55 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[7] = w[4] ^ w[15] ^ w[9] ^ w[7];
    w[7] = w[7] << 1 | w[7] >> 31;

    //56 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[8] = w[5] ^ w[0] ^ w[10] ^ w[8];
    w[8] = w[8] << 1 | w[8] >> 31;

    //57 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[9] = w[6] ^ w[1] ^ w[11] ^ w[9];
    w[9] = w[9] << 1 | w[9] >> 31;

    //58 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[10] = w[7] ^ w[2] ^ w[12] ^ w[10];
    w[10] = w[10] << 1 | w[10] >> 31;

    //59 of 40-60
    f = (b & c) | (b & d) | (c & d);
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[11] = w[8] ^ w[3] ^ w[13] ^ w[11];
    w[11] = w[11] << 1 | w[11] >> 31;

    k = 0xCA62C1D6;

    //60 of 60-64
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[12] = w[9] ^ w[4] ^ w[14] ^ w[12];
    w[12] = w[12] << 1 | w[12] >> 31;

    //61 of 60-64
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[13] = w[10] ^ w[5] ^ w[15] ^ w[13];
    w[13] = w[13] << 1 | w[13] >> 31;

    //62 of 60-64
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[14] = w[11] ^ w[6] ^ w[0] ^ w[14];
    w[14] = w[14] << 1 | w[14] >> 31;

    //63 of 60-64
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    w[15] = w[12] ^ w[7] ^ w[1] ^ w[15];
    w[15] = w[15] << 1 | w[15] >> 31;

    //64 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[0];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //65 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[1];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //66 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[2];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //67 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[3];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //68 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[4];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //69 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[5];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //70 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[6];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //71 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[7];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //72 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[8];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //73 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[9];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //74 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[10];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //75 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[11];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //76 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[12];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //77 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[13];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //78 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[14];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    //79 of 64-80
    f = b ^ c ^ d;
    temp = ((a << 5) | (a >> 27)) + f + e + k + w[15];
    e = d;
    d = c;
    c = (b << 30) | (b >> 2);
    b = a;
    a = temp;

    h.h1 += a;
    h.h2 += b;
    h.h3 += c;
    h.h4 += d;
    h.h5 += e;

}
/*
 __global__ void computeSHA1(char* buf, int *offsets, int *len, char* output, int N)
 {

 //__shared__ uint32_t w_shared[16*SHA1_THREADS_PER_BLK];
 uint32_t w_register[16];

 int index = blockIdx.x * blockDim.x + threadIdx.x;
 if (index < N) {
 uint32_t *w = w_register;//w_shared + 16*threadIdx.x;
 hash_digest_t h;
 h.h1 = 0x67452301;
 h.h2 = 0xEFCDAB89;
 h.h3 = 0x98BADCFE;
 h.h4 = 0x10325476;
 h.h5 = 0xC3D2E1F0;

 int num_iter = (len[index]+63+9)/64;
 printf("num_iter %d\n", num_iter);
 for(int i = 0; i < num_iter; i++)
 computeSHA1Block(buf + offsets[index], w, i*64 , len[index], h);

 h.h1 = swap(h.h1);
 h.h2 = swap(h.h2);
 h.h3 = swap(h.h3);
 h.h4 = swap(h.h4);
 h.h5 = swap(h.h5);

 uint32_t * out = (uint32_t*)(output + index*20);
 *(out++) = h.h1;
 *(out++) = h.h2;
 *(out++) = h.h3;
 *(out++) = h.h4;
 *(out++) = h.h5;
 }
 }*/
/*
 some how *pad = *pad++ ^ *key++
 was optimized and does not work correctly in GPU oTL.
 */
__device__ static void xorpads(uint32_t *pad, const uint32_t* key) {
#pragma unroll 16
    for (int i = 0; i < 16; i++)
        *(pad + i) = *(pad + i) ^ *(key + i);
}

uint32_t opad[16] =
        { 0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c,
          0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c,
          0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c, 0x5c5c5c5c,
          0x5c5c5c5c, };
uint32_t ipad[16] =
        { 0x36363636, 0x36363636, 0x36363636, 0x36363636, 0x36363636,
          0x36363636, 0x36363636, 0x36363636, 0x36363636, 0x36363636,
          0x36363636, 0x36363636, 0x36363636, 0x36363636, 0x36363636,
          0x36363636, };

// in: start pointer of the data to be authenticated by hsha1.
// out: start pointer of the data where hsha1 signature will be recorded.
// length: length of the data to be authenticated by hsha1.
// key: hmac key.
__device__ static void HMAC_SHA1(uint32_t *in, uint32_t *out, uint32_t length,
        const char *key) {
    uint32_t w_register[16];

    uint32_t *w = w_register; //w_shared + 16*threadIdx.x;
    hash_digest_t h;

    for (int i = 0; i < 16; i++)
        w[i] = 0x36363636;
    xorpads(w, (uint32_t*) (key));

    h.h1 = 0x67452301;
    h.h2 = 0xEFCDAB89;
    h.h3 = 0x98BADCFE;
    h.h4 = 0x10325476;
    h.h5 = 0xC3D2E1F0;

    //SHA1 compute on ipad
    computeSHA1Block((char*) w, w, 0, 64, h);

    //SHA1 compute on mesage
    int num_iter = (length + 63 + 9) / 64;
    for (int i = 0; i < num_iter; i++)
        computeSHA1Block((char*) in, w, i * 64, length, h);

    *(out) = swap(h.h1);
    *(out + 1) = swap(h.h2);
    *(out + 2) = swap(h.h3);
    *(out + 3) = swap(h.h4);
    *(out + 4) = swap(h.h5);

    h.h1 = 0x67452301;
    h.h2 = 0xEFCDAB89;
    h.h3 = 0x98BADCFE;
    h.h4 = 0x10325476;
    h.h5 = 0xC3D2E1F0;

    for (int i = 0; i < 16; i++)
        w[i] = 0x5c5c5c5c;

    xorpads(w, (uint32_t*) (key));

    //SHA 1 compute on opads
    computeSHA1Block((char*) w, w, 0, 64, h);

    //SHA 1 compute on (hash of ipad|m)
    computeSHA1Block((char*) out, w, 0, 20, h);

    *(out) = swap(h.h1);
    *(out + 1) = swap(h.h2);
    *(out + 2) = swap(h.h3);
    *(out + 3) = swap(h.h4);
    *(out + 4) = swap(h.h5);
}
#if 0
__global__ void computeHMAC_SHA1(char* buf, char* keys, uint32_t *offsets,
                                 uint32_t *lengths, uint32_t *outputs, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        printf("index%d threadid%d\n", index, threadIdx.x);
        uint32_t offset = offsets[index];
        uint32_t length = lengths[index];
        uint32_t *out = (uint32_t*) (buf + outputs[index]);
        HMAC_SHA1((uint32_t*) (buf + offset), out, length, keys + 64 * index);
    }
}

__global__ void computeHMAC_SHA1_2(char* buf, char* keys, uint32_t *offsets,
                                   uint16_t *lengths, int N) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        printf("index%d threadid%d\n", index, threadIdx.x);
        uint32_t offset = offsets[index];
        uint32_t length = lengths[index];
        uint32_t *out = (uint32_t*) (buf + offset + length);
        HMAC_SHA1((uint32_t*) (buf + offset), out, length, keys + 64 * index);
    }
}
#endif

__global__ void computeHMAC_SHA1_3(
        struct datablock_kernel_arg **datablocks,
        uint32_t count, uint32_t *item_counts, uint32_t num_batches,
        uint8_t *checkbits_d,
        struct hmac_sa_entry *hmac_key_array)
{
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < count && count != 0) {
        uint32_t batch_idx, item_idx;
        nba::error_t err;
        err = nba::get_accum_idx(item_counts, num_batches, idx, batch_idx, item_idx);
        assert(err == nba::NBA_SUCCESS);

        const struct datablock_kernel_arg *db_enc_payloads = datablocks[dbid_enc_payloads_d];
        const struct datablock_kernel_arg *db_flow_ids     = datablocks[dbid_flow_ids_d];

        const uint8_t *enc_payload_base = (uint8_t *) db_enc_payloads->batches[batch_idx].buffer_bases;
        const uintptr_t offset = (uintptr_t) db_enc_payloads->batches[batch_idx].item_offsets[item_idx].as_value<uintptr_t>();
        const uintptr_t length = (uintptr_t) db_enc_payloads->batches[batch_idx].item_sizes[item_idx];
        if (enc_payload_base != NULL && length != 0) {
            const uint64_t flow_id = ((uint64_t *) db_flow_ids->batches[batch_idx].buffer_bases)[item_idx];
            if (flow_id != 65536) {
                assert(flow_id < 1024);
                const char *hmac_key = (char *) hmac_key_array[flow_id].hmac_key;
                HMAC_SHA1((uint32_t *) (enc_payload_base + offset),
                          (uint32_t *) (enc_payload_base + offset + length),
                          length, hmac_key);
            }
        }

        __syncthreads();
        if (threadIdx.x == 0 && checkbits_d != NULL)
            checkbits_d[blockIdx.x] = 1;
    } // endif(valid-idx)
}

}

void *nba::ipsec_hsha1_encryption_get_cuda_kernel() {
    return reinterpret_cast<void *> (computeHMAC_SHA1_3);
}

// vim: ts=8 sts=4 sw=4 et tw=150
